#include "hip/hip_runtime.h"
#include <stdio.h>
#include <math.h>
#include <windows.h>
#include <wchar.h>


#define NUM_PARTICLES 10000
#define NUM_ITERATIONS 100
#define BLOCK_SIZE 256

typedef struct 
{
    float3 Position;
    float3 Velocity;
} Particle;

typedef struct
{
    int block_size;
    int num_iterations;
    int num_particles;
} Info;

typedef struct 
{
    Particle *particles;
    Particle *d_particles;
    Particle *cuda_particles;
} ParticleCollection;



__global__
void timestep_for_gpu(Particle *particles, int num_particles, int iteration)
{
    const float dt = 1.0f;
    const int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < num_particles)
    {
       particles[i].Velocity.x = (iteration % 3 == 0) ? (particles[i].Velocity.x - 0.01) : (particles[i].Velocity.x + 0.01);                                    //Update velocity for the particle
       particles[i].Velocity.y = (iteration % 4 == 0) ? (particles[i].Velocity.y - 0.01) : (particles[i].Velocity.y + 0.01);
       particles[i].Velocity.z = (iteration % 5 == 0) ? (particles[i].Velocity.z - 0.01) : (particles[i].Velocity.z + 0.01);

       particles[i].Position.x += particles[i].Velocity.x * dt;        //Update the position of the particle based on the updated velocity
       particles[i].Position.y += particles[i].Velocity.y * dt;
       particles[i].Position.z += particles[i].Velocity.z * dt;
    }
}

void timestep_for_cpu(Particle *particles, int num_particles, int iteration)
{
    const float dt = 1.0f;
    for(int i = 0; i < num_particles; ++i)
    {
       particles[i].Velocity.x = (iteration % 3 == 0) ? (particles[i].Velocity.x - 0.01) : (particles[i].Velocity.x + 0.01);                                    //Update velocity for the particle
       particles[i].Velocity.y = (iteration % 4 == 0) ? (particles[i].Velocity.y - 0.01) : (particles[i].Velocity.y + 0.01);
       particles[i].Velocity.z = (iteration % 5 == 0) ? (particles[i].Velocity.z - 0.01) : (particles[i].Velocity.z + 0.01);

       particles[i].Position.x += particles[i].Velocity.x * dt;        //Update the position of the particle based on the updated velocity
       particles[i].Position.y += particles[i].Velocity.y * dt;
       particles[i].Position.z += particles[i].Velocity.z * dt;
    }
    return;
}

void init(Info *info, ParticleCollection *particleCollection)
{
    // particleCollection->particles = (Particle*)malloc(info->num_particles * sizeof(Particle));            //Allocate the space needed on the host
    hipHostAlloc((void **) &(particleCollection->particles), info->num_particles * sizeof(Particle), hipHostMallocDefault);
    particleCollection->cuda_particles = (Particle*)malloc(info->num_particles * sizeof(Particle));

    for(int i = 0; i < info->num_particles; ++i)
    {
       particleCollection->particles[i].Velocity.x = 0.7f;
       particleCollection->particles[i].Velocity.y = 0.6f;
       particleCollection->particles[i].Velocity.z = 0.5f;
       particleCollection->particles[i].Position.x = 0;
       particleCollection->particles[i].Position.y = 0;
       particleCollection->particles[i].Position.z = 0;
    }

    hipMalloc(&(particleCollection->d_particles), info->num_particles * sizeof(Particle));             //Allocate the space needed on the device
    //hipMemcpy(particleCollection->d_particles, particleCollection->particles, info->num_particles * sizeof(Particle), hipMemcpyHostToDevice);   //Copy the arrays from the the host to the device
    return;
}

void cpu_execution(Particle *particles, Info *info)
{
    clock_t starting_time = clock();
    for(int iteration = 0; iteration < info->num_iterations; ++iteration)
    {
        timestep_for_cpu(particles, info->num_particles, iteration);
    }
    //printf("Done with the simulation on the CPU, using %d iterations and %d particles. It took %lf seconds to do!\n", info->num_iterations, info->num_particles, (double) (clock() - starting_time) / CLOCKS_PER_SEC);
    return;

}

void gpu_execution(ParticleCollection *particleCollection, Info *info)
{
    int grid_size = (info->num_particles + info->block_size - 1) / info->block_size;
    clock_t starting_time = clock();            //Do the calculations on the GPU
    for(int iteration = 0; iteration < info->num_iterations; ++iteration)
    {
        hipMemcpy(particleCollection->d_particles, particleCollection->particles, info->num_particles * sizeof(Particle), hipMemcpyHostToDevice);
        timestep_for_gpu<<<grid_size, info->block_size>>>(particleCollection->d_particles, info->num_particles, iteration);
        hipDeviceSynchronize();
        hipMemcpy(particleCollection->particles, particleCollection->d_particles, info->num_particles * sizeof(Particle), hipMemcpyDeviceToHost);
    }
    //printf("Done with the simulation on the GPU, using %d iterations and %d particles. It took %lf seconds to do!\n", info->num_iterations, info->num_particles, (double) (clock() - starting_time) / CLOCKS_PER_SEC);
    return;
}

int main(int argc, char *argv[])
{
    Info *info = (Info*) malloc(sizeof(Info));
    ParticleCollection *particleCollection = (ParticleCollection*) malloc(sizeof(particleCollection));
    int correctness = 1;

    info->block_size = argc > 1 ? atoi(argv[1]) : BLOCK_SIZE;
    info->num_iterations = argc > 2 ? atoi(argv[2]) : NUM_ITERATIONS;
    info->num_particles = argc > 3 ? atoi(argv[3]) : NUM_PARTICLES;
    
    init(info, particleCollection);
    
    cpu_execution(particleCollection->particles, info);

    gpu_execution(particleCollection, info);

    hipMemcpy(particleCollection->cuda_particles, particleCollection->d_particles, info->num_particles * sizeof(Particle), hipMemcpyDeviceToHost);

    for(int j = 0; j < info->num_particles; ++j)      //Check if both versions got the same results
    {
        if(particleCollection->particles[j].Position.x != particleCollection->cuda_particles[j].Position.x || 
           particleCollection->particles[j].Position.y != particleCollection->cuda_particles[j].Position.y || 
           particleCollection->particles[j].Position.z != particleCollection->cuda_particles[j].Position.z)
        {
            correctness = 0;
            printf("GPU and CPU does not match at index %d\n", j);
        }
    }
    
    if(correctness == 1)
    {
     //   printf("Comparing the output for each implementation... Correct!\n");
    }
    else
    {
        
        printf("Comparing the output for each implementation... Incorrect!\n");
    }

    // free(particleCollection->particles);
    free(particleCollection->cuda_particles);
    free(info);
    free(particleCollection);
    hipFree(particleCollection->d_particles);
    //printf("Freed\n");
    return 0;
}